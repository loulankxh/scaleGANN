#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <iostream>
#include <vector>
#include <cassert>
#include <omp.h>

#include <thread>
#include <chrono>

#include "merge.cuh"
#include "../taskScheduler/gpuManagement.h"

__global__ void translateShardKernel(
    uint32_t* translated_data,
    uint32_t* index_data,
    uint32_t* idx_vec_data,
    uint32_t neighbor_K,
    uint32_t shard_size) {

    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    uint32_t stride = blockDim.x * gridDim.x;
    for (uint32_t i = idx; i < shard_size; i += stride) {
        if (i >= shard_size) return;

        uint32_t global_index = idx_vec_data[i];

        uint32_t* neighbor_list = &index_data[i * neighbor_K];

        uint32_t offset = i * (neighbor_K + 1);
        translated_data[offset] = global_index; 
        for (uint32_t j = 0; j < neighbor_K; j++) {
            uint32_t neighbor_local_index = neighbor_list[j];
            // if (neighbor_local_index >= shard_size) return;
            translated_data[offset+j+1] = idx_vec_data[neighbor_local_index];
        }
    }

    // __syncthreads();
}


void translateShardGPU(std::vector<std::vector<uint32_t>>& translated_index,
                                   std::vector<std::vector<uint32_t>>& index,
                                   std::vector<uint32_t>& idx_vec,
                                   uint32_t gpu_id) {

    uint32_t shard_size = index.size();
    assert(shard_size == idx_vec.size());
    uint32_t neighbor_K = index[0].size();

    // hipStream_t stream;
    // hipStreamCreate(&stream); // instead of using default CUDA stream, which will block host CPU

    omp_set_lock(&gpu_locks[gpu_id]);

    hipSetDevice(gpu_id);
    if (isGPUBusy(gpu_id)) {
        std::cerr << "GPU " << gpu_id << " is still busy. Waiting..." << std::endl;
        hipEventSynchronize(gpu_end_events[gpu_id]);
    } else{
        std::cerr << "GPU " << gpu_id << " is ready for use" << std::endl;
    }

    hipEventRecord(gpu_start_events[gpu_id], 0);
    // hipEventRecord(gpu_start_events[gpu_id], stream);

    auto hostToDevice_Start = std::chrono::high_resolution_clock::now();

    thrust::device_vector<uint32_t> d_translated_data(shard_size * (neighbor_K + 1));
    thrust::device_vector<uint32_t> d_index_data(shard_size * neighbor_K);
    thrust::device_vector<uint32_t> d_idx_vec_data(shard_size);
    for (uint32_t i = 0; i < shard_size; ++i){
        hipMemcpyAsync(thrust::raw_pointer_cast(d_index_data.data()) + i * neighbor_K, index[i].data(), neighbor_K * sizeof(uint32_t), hipMemcpyHostToDevice);
    }
    hipMemcpyAsync(thrust::raw_pointer_cast(d_idx_vec_data.data()), idx_vec.data(), shard_size * sizeof(uint32_t), hipMemcpyHostToDevice);

    auto hostToDevice_End = std::chrono::high_resolution_clock::now();
    auto hostToDeviceDuration = std::chrono::duration_cast<std::chrono::milliseconds>(hostToDevice_Start - hostToDevice_End);
    printf("Host to Device Transfer time is: %lld ms\n", hostToDeviceDuration.count());
    
    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, gpu_id);
    int threads = properties.maxThreadsPerBlock;
    int maxblocks = properties.multiProcessorCount * min(
        properties.maxThreadsPerMultiProcessor / threads,
        properties.maxBlocksPerMultiProcessor);
    int blocks = min(
        (int)((shard_size + threads - 1) / threads),
        maxblocks);

    translateShardKernel<<<blocks, threads>>>(
        thrust::raw_pointer_cast(d_translated_data.data()),
        thrust::raw_pointer_cast(d_index_data.data()),
        thrust::raw_pointer_cast(d_idx_vec_data.data()),
        neighbor_K,
        shard_size
    );

    auto deviceToHost_Start = std::chrono::high_resolution_clock::now();

    uint32_t row_size = neighbor_K + 1;
    translated_index.resize(shard_size);
    for (uint32_t i = 0; i < shard_size; ++i){
        translated_index[i].resize(row_size);
        hipMemcpyAsync(translated_index[i].data(), thrust::raw_pointer_cast(d_translated_data.data()) + i * row_size, row_size * sizeof(uint32_t), hipMemcpyDeviceToHost);
    }

    auto deviceToHost_End = std::chrono::high_resolution_clock::now();
    auto deviceToHostDuration = std::chrono::duration_cast<std::chrono::milliseconds>(deviceToHost_Start - deviceToHost_End);
    printf("Device to Host Transfer time is: %lld ms\n", deviceToHostDuration.count());

    hipEventRecord(gpu_end_events[gpu_id], 0);
    hipDeviceSynchronize();

    omp_unset_lock(&gpu_locks[gpu_id]);


}