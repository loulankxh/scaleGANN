#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <stdint.h>
#include <cfloat>
#include <vector>
#include <iostream>
#include <cassert>
#include <limits>
#include "kmeans.cuh"


// Lan: why a center may not be the best for every data point ?????
template <typename T>
__global__ void kMeansCUDAKernel(T* data, float* centroids, uint32_t* labels, 
            float* new_centroids, float* residual_, uint32_t* count, 
            uint32_t npts, uint32_t ndim, uint32_t partition_num) {
    int tid = threadIdx.x;
    int bid = blockIdx.x;

    uint32_t idx = bid * blockDim.x + tid;
    // if data num is larger than thread num
    for (uint32_t i = idx; i < npts; i += blockDim.x * gridDim.x){
        float minDist = FLT_MAX;
        uint32_t bestCentroid = 0;
        for (uint32_t j = 0; j < partition_num; ++j) {
            float dist = 0;
            for (uint32_t d = 0; d < ndim; ++d) {
                float diff = data[i * ndim + d] - centroids[j * ndim + d];
                dist += diff * diff;
            }
            if (dist < minDist) {
                minDist = dist;
                bestCentroid = j;
            }
        }
        labels[i] = bestCentroid;
        residual_[i] = minDist;
    }
    __syncthreads();


    for (uint32_t i = idx; i < npts; i += blockDim.x * gridDim.x){
        uint32_t bestCentroid = labels[i];
        atomicAdd(&count[bestCentroid], 1);
        for (uint32_t d = 0; d < ndim; ++d){
            atomicAdd(&new_centroids[bestCentroid * ndim + d], (float) data[i * ndim + d]);
        }
    }
    __syncthreads();

    if(idx < partition_num){
        for (uint32_t d = 0; d < ndim; ++d){
            centroids[idx * ndim + d] = new_centroids[idx * ndim + d] / count[idx];
            if (centroids[idx * ndim + d] == 0.0) {
                printf("count: %d; idx: %d; dimension: %d; new value: %f\n ", count[idx], idx, d, new_centroids[idx * ndim + d]);
            }
        }
    }
    // __syncthreads();

}

template <typename T>
void kMeansCUDA(uint32_t partition_num, uint32_t ndim, uint32_t max_iters, uint32_t device,
const std::vector<std::vector<T>>& sample, std::vector<std::vector<float>>& centroids){
    printf("Entering GPU for kmeans training...\n");
    uint32_t npts = sample.size();

    thrust::device_vector<T> d_data(npts * ndim);
    thrust::device_vector<float> d_centroids(partition_num * ndim);
    thrust::device_vector<uint32_t> d_labels(npts);
    thrust::device_vector<float> d_residual(npts);
    thrust::device_vector<float> d_new_centroids(partition_num * ndim);
    thrust::device_vector<uint32_t> d_count(partition_num);

    static_assert(std::is_arithmetic<T>::value, "T must be a numeric type");
    for (uint32_t i = 0; i < npts; ++i)
        hipMemcpy(thrust::raw_pointer_cast(d_data.data()) + i * ndim, sample[i].data(), ndim * sizeof(T), hipMemcpyHostToDevice);
    for (uint32_t i = 0; i < partition_num; ++i)
        hipMemcpy(thrust::raw_pointer_cast(d_centroids.data()) + i * ndim, centroids[i].data(), ndim * sizeof(float), hipMemcpyHostToDevice);
    
    thrust::fill(d_residual.begin(), d_residual.end(), 0.0);
    thrust::fill(d_count.begin(), d_count.end(), 1);
    thrust::fill(d_new_centroids.begin(), d_new_centroids.end(), 0.0f);
    
    // setting CUDA parameters
    hipSetDevice(device);
    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, device);
    int threads = properties.maxThreadsPerBlock;
    int maxblocks = properties.multiProcessorCount * min(
        properties.maxThreadsPerMultiProcessor / threads,
        properties.maxBlocksPerMultiProcessor);
    int blocks = min(
        (int)((npts + threads - 1) / threads),
        maxblocks);

    // using residual for early stop
    std::vector<float> residual_host(npts);
    float residual = FLT_MAX;
    float oldresidual = FLT_MAX;

    for (int iter = 0; iter < max_iters; ++iter) {
        if(iter != 0){
            assert((residual > 0) && "residual must be greater than 0 except the first iteration");
            if((((oldresidual - residual) / residual) < 0.00001) || (residual < std::numeric_limits<float>::epsilon())){
                break;
            }
        }
        printf("Kmeans iteration %d using GPU\n", iter);

        oldresidual = residual;
        residual = 0.0;

        kMeansCUDAKernel<T><<<blocks, threads>>>(thrust::raw_pointer_cast(d_data.data()), thrust::raw_pointer_cast(d_centroids.data()), 
                            thrust::raw_pointer_cast(d_labels.data()), thrust::raw_pointer_cast(d_new_centroids.data()), 
                            thrust::raw_pointer_cast(d_residual.data()), thrust::raw_pointer_cast(d_count.data()), 
                            npts, ndim, partition_num);
        hipDeviceSynchronize();

        hipMemcpy(residual_host.data(), thrust::raw_pointer_cast(d_residual.data()), npts * sizeof(float), hipMemcpyDeviceToHost);
        for(uint32_t i = 0; i < npts; i++){
            residual += residual_host[i];
        }

        thrust::fill(d_residual.begin(), d_residual.end(), 0.0);
        thrust::fill(d_count.begin(), d_count.end(), 1);
        thrust::fill(d_new_centroids.begin(), d_new_centroids.end(), 0);
    }
    
    for (uint32_t i = 0; i < partition_num; ++i){
        hipMemcpy(centroids[i].data(), thrust::raw_pointer_cast(d_centroids.data()) + i * ndim, ndim * sizeof(float), hipMemcpyDeviceToHost);
    }
}



template void kMeansCUDA<float>(uint32_t partition_num, uint32_t ndim, uint32_t max_iters, uint32_t device,
const std::vector<std::vector<float>>& sample, std::vector<std::vector<float>>& centroids);
template void kMeansCUDA<uint32_t>(uint32_t partition_num, uint32_t ndim, uint32_t max_iters, uint32_t device,
const std::vector<std::vector<uint32_t>>& sample, std::vector<std::vector<float>>& centroids);
template void kMeansCUDA<uint8_t>(uint32_t partition_num, uint32_t ndim, uint32_t max_iters, uint32_t device,
const std::vector<std::vector<uint8_t>>& sample, std::vector<std::vector<float>>& centroids);